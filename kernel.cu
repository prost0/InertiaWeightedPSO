#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <GL/glew.h>
#include <GL/freeglut.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#pragma comment(lib, "glew32.lib")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <chrono>

using namespace std;

#define CSC(call) {							\
    hipError_t err = call;						\
    if(err != hipSuccess) {						\
        fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n",	\
            __FILE__, __LINE__, hipGetErrorString(err));		\
        exit(1);							\
    }									\
} while (0)

#define square(x) ((x)*(x))
#define THREADS 128

struct particle
{
	double2 coord;
	double2 velocity;
	double2 best_coord;
	double2 repultion_force;
};


const int width = 1280;
const int height = 720;

const int particle_cnt = 4000;
const double inertia = 0.981;
const double coef_local = 0.4;
const double coef_global = 0.15;
const double coef_repultion = 0.5;
const double dt = 0.07;

double scale_x = 500;
double scale_y = scale_x * height / width;

const dim3 blocks2D(128, 128);
const dim3 threads2D(32, 32);
const int threads_reduce = 1024;
const int blocks_reduce = width * height / threads_reduce + 1;
const int threads1D = THREADS;
const int blocks1D = ceil((double)particle_cnt / THREADS);

__constant__ double pi = 3.1415;
__constant__ int seed = 1234;

__device__ double dev_center_x = 0;
__device__ double dev_center_y = 0;
__device__ double dev_func_min;
__device__ double dev_func_max;
__device__ double image[height * width];
__device__ double2 g_best;

hiprandState* dev_states;

struct hipGraphicsResource *res;
particle *dev_swarm;


double *arr_max_after_reduce_dev;
double *arr_min_after_reduce_dev;

double2 *global_best_after_reduce;

GLuint vbo;


__device__ double rosenbrock(double2 arg) {
	return square((1 - arg.x)) + 100 * square((arg.y - square(arg.x)));
}


__device__ double rosenbrock(int i, int j, double scale_x, double scale_y) {
	double x = 2.0f * i / (double)(width - 1) - 1.0f;
	double y = 2.0f * j / (double)(height - 1) - 1.0f;
	return rosenbrock(make_double2(x * scale_x + dev_center_x, -y * scale_y + dev_center_y));
}


__global__ void rosenbrock_image(double scale_x, double scale_y) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;

	for (int j = idy; j < height; j += offsety)
	{
		for (int i = idx; i < width; i += offsetx)
		{
			image[j * width + i] = rosenbrock(i, j, scale_x, scale_y);
		}
	}
}


__global__ void minmax_reduce(double *arr_min_after_reduce, double *arr_max_after_reduce)
{
	__shared__ double shared_min[threads_reduce];
	__shared__ double shared_max[threads_reduce];

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < width * height)
	{
		shared_min[threadIdx.x] = image[idx];
		shared_max[threadIdx.x] = image[idx];
	}
	else
	{
		shared_min[threadIdx.x] = INFINITY;
		shared_max[threadIdx.x] = -INFINITY;
	}
	__syncthreads();

	for (int step = 2; step <= threads_reduce; step *= 2)
	{
		if (threadIdx.x * (step + 1) - 1 < threads_reduce)
		{
			shared_min[threadIdx.x * (step + 1) - 1] = (shared_min[threadIdx.x * (step + 1) - 1] < shared_min[threadIdx.x * (step + 1) - step / 2 - 1]) ? shared_min[threadIdx.x * (step + 1) - 1] : shared_min[threadIdx.x * (step + 1) - step / 2 - 1];
			shared_max[threadIdx.x * (step + 1) - 1] = (shared_max[threadIdx.x * (step + 1) - 1] > shared_max[threadIdx.x * (step + 1) - step / 2 - 1]) ? shared_max[threadIdx.x * (step + 1) - 1] : shared_max[threadIdx.x * (step + 1) - step / 2 - 1];
		}
		__syncthreads();
	}

	if (threadIdx.x == 0)
	{
		arr_min_after_reduce[blockIdx.x] = shared_min[threads_reduce - 1];
		arr_max_after_reduce[blockIdx.x] = shared_max[threads_reduce - 1];
	}

}

__global__ void minmax(double *arr_min_after_reduce, double *arr_max_after_reduce, int size)
{
	double min = arr_min_after_reduce[0];
	double max = arr_max_after_reduce[0];
	for (int i = 1; i < size; i++)
	{
		if (arr_min_after_reduce[i] < arr_min_after_reduce[i - 1])
			min = arr_min_after_reduce[i];
		if (arr_max_after_reduce[i] > arr_max_after_reduce[i - 1])
			max = arr_max_after_reduce[i];
	}
	dev_func_min = min;
	dev_func_max = max;
}

__device__ uchar4 get_color(double f) {
	float k = 1.0 / 6.0;
	if (f <= 0)
		return make_uchar4(0, 0, 0, 0);
	if (f < k)
		return make_uchar4((int)(f * 255 / k), 0, 0, 0);
	if (f < 2 * k)
		return make_uchar4(255, (int)((f - k) * 255 / k), 0, 0);
	if (f < 3 * k)
		return make_uchar4(255, 255, (int)((f - 2 * k) * 255 / k), 0);
	if (f < 4 * k)
		return make_uchar4(255 - (int)((f - 3 * k) * 255 / k), 255, 255, 0);
	if (f < 5 * k)
		return make_uchar4(0, 255 - (int)((f - 4 * k) * 255 / k), 255, 0);
	if (f <= 6 * k)
		return make_uchar4(0, 0, 255 - (int)((f - 5 * k) * 255 / k), 0);
	return make_uchar4(0, 0, 0, 0);
}

__global__ void heatmap(uchar4* data)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;

	for (int j = idy; j < height; j += offsety)
	{
		for (int i = idx; i < width; i += offsetx)
		{
			data[j * width + i] = get_color((image[j * width + i] - dev_func_min) / (dev_func_max - dev_func_min));
		}
	}
}


__global__ void update_coords_and_velocities(double inertia, double coef_local, double coef_global, double dt, double coef_repultion,
	particle *swarm, int particle_cnt, uchar4* data, double scale_x, double scale_y, hiprandState * state)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int offsetx = blockDim.x * gridDim.x;

	for (int i = idx; i < particle_cnt; i += offsetx)
	{
		swarm[idx].velocity.x = inertia * swarm[idx].velocity.x + (coef_local * hiprand_uniform(&state[idx]) * (swarm[idx].best_coord.x - swarm[idx].coord.x) +
			coef_global * hiprand_uniform(state) * (g_best.x - swarm[idx].coord.x) + coef_repultion * swarm[idx].repultion_force.x) * dt;
		swarm[idx].velocity.y = inertia * swarm[idx].velocity.y + (coef_local * hiprand_uniform(&state[idx]) * (swarm[idx].best_coord.y - swarm[idx].coord.y) +
			coef_global * hiprand_uniform(state) * (g_best.y - swarm[idx].coord.y) + coef_repultion * swarm[idx].repultion_force.y) * dt;
		swarm[idx].coord.x += swarm[idx].velocity.x * dt;
		swarm[idx].coord.y += swarm[idx].velocity.y * dt;
		if (rosenbrock(make_double2(swarm[idx].coord.x, swarm[idx].coord.y)) < rosenbrock(make_double2(swarm[idx].best_coord.x, swarm[idx].best_coord.y)))
		{
			swarm[idx].best_coord.x = swarm[idx].coord.x;
			swarm[idx].best_coord.y = swarm[idx].coord.y;
		}
		double2 particle_draw_coord;
		particle_draw_coord.x = (((swarm[idx].coord.x - dev_center_x) / scale_x) + 1) * (width - 1) / 2;
		particle_draw_coord.y = (1 - ((swarm[idx].coord.y - dev_center_y) / scale_y)) * (height - 1) / 2;

		if (particle_draw_coord.x > 0 && particle_draw_coord.x < width && particle_draw_coord.y > 0 && particle_draw_coord.y < height)
		{
			data[(int)particle_draw_coord.y * width + (int)particle_draw_coord.x] = make_uchar4(255, 255, 255, 255);
		}
	}
}

__global__ void repulsive_force(particle *swarm, int particle_cnt)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int offsetx = blockDim.x * gridDim.x;
	double square_dist;

	for (int i = idx; i < particle_cnt; i += offsetx)
	{
		for (int i = 0; i < particle_cnt; i++)
		{
			square_dist = square(swarm[i].coord.x - swarm[idx].coord.x) + square(swarm[i].coord.y - swarm[idx].coord.y);
			swarm[idx].repultion_force.x -= (swarm[i].coord.x - swarm[idx].coord.x) / (square(square_dist) + 1e-3);
			swarm[idx].repultion_force.y -= (swarm[i].coord.y - swarm[idx].coord.y) / (square(square_dist) + 1e-3);
		}
	}
}

__global__ void update_window_center(particle *swarm, int particle_cnt)
{
	double2 sum;
	for (int i = 0; i < particle_cnt; i++)
	{
		sum.x += swarm[i].coord.x;
		sum.y += swarm[i].coord.y;
	}
	sum.x /= particle_cnt;
	sum.y /= particle_cnt;
	dev_center_x = sum.x;
	dev_center_y = sum.y;
}


__global__ void global_best_reduce(particle *swarm, double2 *global_best_after_reduce)
{
	__shared__ double2 shared_min[threads_reduce];
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	shared_min[threadIdx.x] = swarm[idx].coord;
	__syncthreads();

	for (int step = 2; step <= threads_reduce; step *= 2)
	{
		if (threadIdx.x * (step + 1) - 1 < threads_reduce)
		{
			shared_min[threadIdx.x * (step + 1) - 1] = (rosenbrock(shared_min[threadIdx.x * (step + 1) - 1]) < rosenbrock(shared_min[threadIdx.x * (step + 1) - step / 2 - 1])) ?
				shared_min[threadIdx.x * (step + 1) - 1] : shared_min[threadIdx.x * (step + 1) - step / 2 - 1];
		}
		__syncthreads();
		step *= 2;
	}
	if (threadIdx.x == 0)
	{
		global_best_after_reduce[blockIdx.x] = shared_min[threads_reduce - 1];
	}
}

__global__ void global_best_final(particle *swarm, double2 *global_best_after_reduce, int size)
{
	double2 max;
	if (size > 0)
	{
		max = global_best_after_reduce[0];
	}
	else
	{
		max = swarm[0].coord;
	}
	for (int i = 1; i < size; i++)
	{
		if (rosenbrock(global_best_after_reduce[i]) < rosenbrock(global_best_after_reduce[i - 1]))
			max = global_best_after_reduce[i];
	}
	for (int i = THREADS * size; i < size; i++)
	{
		if (rosenbrock(swarm[i].coord) < rosenbrock(max))
			max = swarm[i].coord;
	}
	if (rosenbrock(max) < rosenbrock(g_best))
		g_best = max;
}


//__global__ void global_best_final(particle *swarm, double2 *global_best_after_reduce, int size, int particle_cnt)
//{
//		for (int i = 0; i < size; i++)
//		{
//			if (rosenbrock(global_best_after_reduce[i]) < rosenbrock(g_best))
//				g_best = global_best_after_reduce[i];
//		}
//		for (int i = size; i < size; i++)
//		{
//			if (rosenbrock(swarm[i].coord) < rosenbrock(g_best))
//				g_best = swarm[i].coord;
//		}
//}

__global__ void swarm_start(particle *swarm, int particle_cnt, hiprandState * state)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int offsetx = blockDim.x * gridDim.x;

	for (int i = idx; i < particle_cnt; i += offsetx)
	{
		hiprand_init(seed, idx, 0, &state[idx]);

		swarm[idx].best_coord.x = swarm[idx].coord.x = hiprand_uniform(&state[idx]) * width * cos((double)idx / THREADS * 2 * pi);
		swarm[idx].best_coord.y = swarm[idx].coord.y = hiprand_uniform(&state[idx]) * height * sin((double)idx / THREADS * 2 * pi);

		swarm[idx].velocity = make_double2(0, 0);
		swarm[idx].repultion_force = make_double2(0, 0);

	}
}


void update() {
	uchar4* heat_image;
	size_t size;
	CSC(hipGraphicsMapResources(1, &res, 0));
	CSC(hipGraphicsResourceGetMappedPointer((void**)&heat_image, &size, res));

	float time;
	hipEvent_t start, stop;
	CSC(hipEventCreate(&start));
	CSC(hipEventCreate(&stop));
	CSC(hipEventRecord(start, 0));

	update_window_center << <1, 1 >> > (dev_swarm, particle_cnt);
	CSC(hipGetLastError());

	rosenbrock_image << <blocks2D, threads2D >> > (scale_x, scale_y);
	CSC(hipGetLastError());

	minmax_reduce << <blocks_reduce, threads_reduce >> > (arr_min_after_reduce_dev, arr_max_after_reduce_dev);
	CSC(hipGetLastError());

	minmax << <1, 1 >> > (arr_min_after_reduce_dev, arr_max_after_reduce_dev, blocks_reduce);
	CSC(hipGetLastError());

	heatmap << <blocks2D, threads2D >> > (heat_image);
	CSC(hipGetLastError());

	repulsive_force << <blocks1D, threads1D >> > (dev_swarm, particle_cnt);
	CSC(hipGetLastError());

	update_coords_and_velocities << <blocks1D, threads1D >> > (inertia, coef_local, coef_global, dt, coef_repultion, dev_swarm, particle_cnt, heat_image, scale_x, scale_y, dev_states);
	CSC(hipGetLastError());

	global_best_reduce << <blocks_reduce, threads_reduce >> > (dev_swarm, global_best_after_reduce);
	CSC(hipGetLastError());

	global_best_final << <1, 1 >> > (dev_swarm, global_best_after_reduce, blocks_reduce);
	CSC(hipGetLastError());

	CSC(hipDeviceSynchronize());
	CSC(hipGraphicsUnmapResources(1, &res, 0));

	CSC(hipEventRecord(stop, 0));
	CSC(hipEventSynchronize(stop));
	CSC(hipEventElapsedTime(&time, start, stop));
	CSC(hipEventDestroy(start));
	CSC(hipEventDestroy(stop));
	printf("%.4f\n", time);

	glutPostRedisplay();
}

void display() {
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT);
	glDrawPixels(width, height, GL_RGBA, GL_UNSIGNED_BYTE, 0);
	glutSwapBuffers();
}

void MyKeyboardFunc(unsigned char Key, int x, int y)
{
	switch (Key)
	{
	case 27:
		CSC(hipGraphicsUnregisterResource(res));
		glBindBuffer(1, vbo);
		glDeleteBuffers(1, &vbo);
		exit(0);
		break;
	case 'q':
		scale_x += 20;
		scale_y = scale_x * height / width;
		break;
	case 'e':
		if (scale_x > 30)
		{
			scale_x -= 20;
			scale_y = scale_x * height / width;
		}
		break;
	};
}

int main(int argc, char** argv)
{
	CSC(hipMalloc(&dev_swarm, sizeof(particle) * (ceil(particle_cnt / (double)THREADS))  * THREADS));
	CSC(hipMalloc(&dev_states, sizeof(hiprandState) * (ceil(particle_cnt / (double)THREADS)) * THREADS));
	CSC(hipMalloc(&global_best_after_reduce, sizeof(double2) * ceil(particle_cnt / (double)THREADS)));
	CSC(hipMalloc(&arr_max_after_reduce_dev, sizeof(double) * blocks_reduce));
	CSC(hipMalloc(&arr_min_after_reduce_dev, sizeof(double) * blocks_reduce));

	swarm_start << <blocks1D, threads1D >> > (dev_swarm, particle_cnt, dev_states);
	CSC(hipGetLastError());

	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
	glutInitWindowSize(width, height);
	glutCreateWindow("YakimovichCP");

	glutIdleFunc(update);
	glutDisplayFunc(display);
	glutKeyboardFunc(MyKeyboardFunc);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(0.0, (GLdouble)width, 0.0, (GLdouble)height);

	glewInit();

	glGenBuffers(1, &vbo);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, vbo);
	glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, width * height * sizeof(uchar4), NULL, GL_DYNAMIC_DRAW);

	CSC(hipGraphicsGLRegisterBuffer(&res, vbo, cudaGraphicsMapFlagsWriteDiscard));

	glutMainLoop();

	return 0;
}